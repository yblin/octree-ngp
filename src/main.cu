#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   main.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <filesystem/path.h>

#include "codelibrary/base/log.h"

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

int main_func(const std::vector<std::string>& arguments) {
    LOG_ON(INFO);

	ArgumentParser parser{
		"Instant Neural Graphics Primitives\n"
		"Version " NGP_VERSION,
		"",
	};

	HelpFlag help_flag{
		parser,
		"HELP",
		"Display this help menu.",
		{'h', "help"},
	};

	ValueFlag<string> network_config_flag{
		parser,
		"CONFIG",
		"Path to the network config. Uses the scene's default if unspecified.",
		{'n', 'c', "network", "config"},
	};

	Flag no_gui_flag{
		parser,
		"NO_GUI",
        "Disables the GUI and instead reports training progress on the command "
        "line.",
		{"no-gui"},
	};

    Flag train_flag{
        parser,
        "TRAIN",
        "Train street views.",
        {'t', "train"},
    };

    Flag render_flag{
        parser,
        "RENDER",
        "Render street views.",
        {'r', "render"},
    };

	ValueFlag<string> snapshot_flag{
		parser,
		"SNAPSHOT",
		"Optional snapshot to load upon startup.",
		{"snapshot", "load_snapshot"},
	};

	ValueFlag<uint32_t> width_flag{
		parser,
		"WIDTH",
		"Resolution width of the GUI.",
		{"width"},
	};

	ValueFlag<uint32_t> height_flag{
		parser,
		"HEIGHT",
		"Resolution height of the GUI.",
		{"height"},
	};

	Flag version_flag{
		parser,
		"VERSION",
		"Display the version of instant neural graphics primitives.",
		{'v', "version"},
	};

	PositionalList<string> files{
		parser,
		"files",
        "Files to be loaded. Can be a scene, network config, snapshot, camera "
        "path, or a combination of those.",
	};

	// Parse command line arguments and react to parsing
	// errors using exceptions.
	try {
		if (arguments.empty()) {
			tlog::error() << "Number of arguments must be bigger than 0.";
			return -3;
		}

		parser.Prog(arguments.front());
		parser.ParseArgs(begin(arguments) + 1, end(arguments));
	} catch (const Help&) {
		cout << parser;
		return 0;
	} catch (const ParseError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -1;
	} catch (const ValidationError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -2;
	}

	if (version_flag) {
		tlog::none() << "Instant Neural Graphics Primitives v" NGP_VERSION;
		return 0;
	}

	Testbed testbed;

    if (train_flag) {
        for (auto file : get(files)) {
            testbed.train_street_view_nerf(file);
        }
    } else if (render_flag) {
        for (auto file : get(files)) {
            testbed.render_street_view_nerf(file);
        }
    } else {
        // No train and render flags, do traditional instant-NGP.
        for (auto file : get(files)) {
            testbed.load_file(file);
        }
    }

//	if (snapshot_flag) {
//		testbed.load_snapshot(get(snapshot_flag));
//	} else if (network_config_flag) {
//		testbed.reload_network_from_file(get(network_config_flag));
//	}

    testbed.m_train = false;

#ifdef NGP_GUI
	bool gui = !no_gui_flag;
#else
	bool gui = false;
#endif

	if (gui) {
        testbed.init_window(width_flag  ? get(width_flag)  : 1920,
                            height_flag ? get(height_flag) : 1080);
    }

	// Render/training loop
    while (testbed.frame()) {
        if (!gui) {
            tlog::info() << "iteration=" << testbed.m_training_step << " loss="
                         << testbed.m_loss_scalar.val();
		}
	}

	return 0;
}

NGP_NAMESPACE_END

#ifdef _WIN32
int wmain(int argc, wchar_t* argv[]) {
	SetConsoleOutputCP(CP_UTF8);
#else
int main(int argc, char* argv[]) {
#endif
	try {
		std::vector<std::string> arguments;
		for (int i = 0; i < argc; ++i) {
#ifdef _WIN32
			arguments.emplace_back(ngp::utf16_to_utf8(argv[i]));
#else
			arguments.emplace_back(argv[i]);
#endif
		}

		return ngp::main_func(arguments);
	} catch (const exception& e) {
		tlog::error() << fmt::format("Uncaught exception: {}", e.what());
		return 1;
	}
}
