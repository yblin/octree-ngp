#include <hip/hip_runtime.h>
#include <cstdio>
int main() {
	int count = 0;
	if (hipSuccess != hipGetDeviceCount(&count)) return -1;
	if (count == 0) return -1;
	for (int device = 0; device < count; ++device) {
		hipDeviceProp_t prop;
		if (hipSuccess == hipGetDeviceProperties(&prop, device)) {
			std::printf("%d%d", prop.major, prop.minor);
			if (device < count - 1) std::printf(";");
		}
	}
	return 0;
}
